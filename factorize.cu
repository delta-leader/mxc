#include "hip/hip_runtime.h"

#include <factorize.cuh>
#include <algorithm>
#include <numeric>
#include <tuple>
#include <mkl.h>
#include <cstring>

#include <hip/hip_fp16.h>
#include <iostream>

__global__ void tofloat(const __half * __restrict__ in, float * __restrict__ out, const long long N){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < N)
    out[idx] = __half2float(in[idx]);
}

__global__ void set(float* out, const long long N){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < N)
    out[idx] = 1;//__float2half(in[idx]);
}

__global__ void tohalf(const float * __restrict__ in, __half * __restrict__ out, const long long N){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < N)
    out[idx] = __float2half(in[idx]);
}

/* helper functions for different datatypes */
template <typename DT>
// complex double
inline void fill_zero(DT* start, DT* end) {
  std::fill(start, end, (DT) 0);
}

// complex double
template <>
inline void fill_zero(hipDoubleComplex* start, hipDoubleComplex* end) {
  std::fill(start, end, make_hipDoubleComplex(0., 0.));
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, const std::complex<double> *SRC, size_t src_stride, hipDoubleComplex *DST, size_t dst_stride) {
  MKL_Zomatcopy(ordering, trans, rows, cols, std::complex<double>(1., 0.), SRC, src_stride, reinterpret_cast<std::complex<double>*>(DST), dst_stride);
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, hipDoubleComplex *SRC, size_t src_stride, std::complex<double> *DST, size_t dst_stride) {
  MKL_Zomatcopy(ordering, trans, rows, cols, std::complex<double>(1., 0.), reinterpret_cast<std::complex<double>*>(SRC), src_stride, DST, dst_stride);
}
// complex float
template <>
inline void fill_zero(hipComplex* start, hipComplex* end) {
  std::fill(start, end, make_hipComplex(0., 0.));
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, const std::complex<float> *SRC, size_t src_stride, hipComplex *DST, size_t dst_stride) {
  MKL_Comatcopy(ordering, trans, rows, cols, std::complex<float>(1., 0.), SRC, src_stride, reinterpret_cast<std::complex<float>*>(DST), dst_stride);
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, hipComplex *SRC, size_t src_stride, std::complex<float> *DST, size_t dst_stride) {
  MKL_Comatcopy(ordering, trans, rows, cols, std::complex<float>(1., 0.), reinterpret_cast<std::complex<float>*>(SRC), src_stride, DST, dst_stride);
}
// double
void omatcopy(char ordering, char trans, size_t rows, size_t cols, const double *SRC, size_t src_stride, double *DST, size_t dst_stride) {
  MKL_Domatcopy(ordering, trans, rows, cols, 1, SRC, src_stride, DST, dst_stride);
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, double *SRC, size_t src_stride, double *DST, size_t dst_stride) {
  MKL_Domatcopy(ordering, trans, rows, cols, 1, SRC, src_stride, DST, dst_stride);
}
// float
void omatcopy(char ordering, char trans, size_t rows, size_t cols, const float *SRC, size_t src_stride, float *DST, size_t dst_stride) {
  MKL_Somatcopy(ordering, trans, rows, cols, 1, SRC, src_stride, DST, dst_stride);
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, float *SRC, size_t src_stride, float *DST, size_t dst_stride) {
  MKL_Somatcopy(ordering, trans, rows, cols, 1, SRC, src_stride, DST, dst_stride);
}

template <typename DT>
H2Factorize<DT>::H2Factorize(long long LD, long long lenA, long long lenQ, hipStream_t stream) : maxA(lenA), maxQ(lenQ), bdim(LD), stream(stream) {
  cublasH = nullptr;
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  long long bsize = LD * LD * sizeof(DT);
  hipMalloc(reinterpret_cast<void**>(&Adata), bsize * lenA);
  hipMalloc(reinterpret_cast<void**>(&Bdata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Udata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Vdata), bsize * lenQ);

  long long psize = sizeof(DT*);
  hipMalloc(reinterpret_cast<void**>(&A_SS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_SR), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RR), psize * lenA);

  hipMalloc(reinterpret_cast<void**>(&B), psize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&U), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V_R), psize * lenQ);

  hipMalloc(reinterpret_cast<void**>(&ipiv), LD * lenQ * sizeof(int));
  hipMalloc(reinterpret_cast<void**>(&info), lenQ * sizeof(int));

  DT** hostB;
  hipHostMalloc(reinterpret_cast<void**>(&hostB), psize * lenQ);
  for (long long i = 0; i < lenQ; i++)
    hostB[i] = &Bdata[i * LD * LD];

  hipMemcpy(B, hostB, psize * lenQ, hipMemcpyHostToDevice);
  hipHostFree(hostB);
}

template <typename DT>
H2Factorize<DT>::~H2Factorize() {
  hipFree(Adata);
  hipFree(Bdata);
  hipFree(Udata);
  hipFree(Vdata);
  hipFree(A_SS);
  hipFree(A_SR);
  hipFree(A_RS);
  hipFree(A_RR);
  hipFree(B);
  hipFree(U);
  hipFree(V);
  hipFree(V_R);
  hipFree(ipiv);
  hipFree(info);
}

template <typename DT> template <typename OT>
void H2Factorize<DT>::setData(long long rank, long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], const MatrixDataContainer<OT>& A, const MatrixDataContainer<OT>& Q) {
  long long block = bdim * bdim;
  lenD = M;
  lenA = std::min(maxA, A.nblocks());
  long long lenQ = std::min(maxQ, Q.nblocks());
  H2Factorize::rank = rank;

  DT* hostU, *hostA;
  hipHostMalloc(reinterpret_cast<void**>(&hostU), block * lenQ * sizeof(DT));
  //std::fill(hostU, &hostU[block * lenQ], make_hipDoubleComplex(0., 0.));
  fill_zero(hostU, &hostU[block * lenQ]);

  for (long long i = 0; i < lenQ; i++) {
    long long m = Dims[i];
    //MKL_Zomatcopy('C', 'C', m, m, std::complex<double>(1., 0.), Q[i], m, reinterpret_cast<std::complex<double>*>(&hostU[i * block]), bdim);
    omatcopy('C', 'C', m, m, Q[i], m, &hostU[i * block], bdim);
  }
  hipMemcpy(Udata, hostU, block * lenQ * sizeof(DT), hipMemcpyHostToDevice);
  hipMemcpy(Vdata, Udata, block * lenQ * sizeof(DT), hipMemcpyDeviceToDevice);
  
  hipHostFree(hostU);
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(DT));
  //std::fill(hostA, &hostA[block * lenA], make_hipDoubleComplex(0., 0.));
  fill_zero(hostA, &hostA[block * lenA]);

  std::vector<std::tuple<long long, long long, long long>> coo_list(lenA);
  for (long long y = 0; y < M; y++) {
    long long begin = ARows[y];
    long long end = ARows[y + 1];
    std::transform(&ACols[begin], &ACols[end], coo_list.begin() + begin, 
      [&](const long long& x) { return std::make_tuple(y + D, x, std::distance(ACols, &x)); });
  }

  for (long long i = 0; i < lenA; i++) {
    long long y = std::get<0>(coo_list[i]);
    long long x = std::get<1>(coo_list[i]);
    long long M = Dims[y], N = Dims[x];
    //MKL_Zomatcopy('C', 'N', M, N, std::complex<double>(1., 0.), A[i], M, reinterpret_cast<std::complex<double>*>(&hostA[i * block]), bdim);
    omatcopy('C', 'N', M, N, A[i], M, &hostA[i * block], bdim);
  }

  hipMemcpy(Adata, hostA, block * lenA * sizeof(DT), hipMemcpyHostToDevice);
  hipHostFree(hostA);

  std::stable_partition(coo_list.begin(), coo_list.end(), 
    [](std::tuple<int64_t, int64_t, int64_t> i) { return std::get<0>(i) == std::get<1>(i); });

  DT** hostAptrs, **hostUptrs, **hostVptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostAptrs), lenA * sizeof(DT*));
  hipHostMalloc(reinterpret_cast<void**>(&hostUptrs), lenA * sizeof(DT*));
  hipHostMalloc(reinterpret_cast<void**>(&hostVptrs), lenA * sizeof(DT*));

  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Udata + std::get<1>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block; });

  hipMemcpy(A_SS, hostAptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(U, hostUptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(V, hostVptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);

  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_SR; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RR; });

  hipMemcpy(A_SR, hostAptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS, hostUptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR, hostVptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);

  std::transform(coo_list.begin(), coo_list.begin() + M, hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block + offset_RS; });
  hipMemcpy(V_R, hostVptrs, M * sizeof(DT*), hipMemcpyHostToDevice);

  hipHostFree(hostAptrs);
  hipHostFree(hostUptrs);
  hipHostFree(hostVptrs);
}

// specialization for experimental purposes
template <> template<>
void H2Factorize<float>::setData(long long rank, long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], const MatrixDataContainer<float>& A, const MatrixDataContainer<float>& Q) {
  long long block = bdim * bdim;
  lenD = M;
  lenA = std::min(maxA, A.nblocks());
  long long lenQ = std::min(maxQ, Q.nblocks());
  H2Factorize::rank = rank;

  float* hostU, *hostA;
  hipHostMalloc(reinterpret_cast<void**>(&hostU), block * lenQ * sizeof(float));
  //std::fill(hostU, &hostU[block * lenQ], make_hipDoubleComplex(0., 0.));
  fill_zero(hostU, &hostU[block * lenQ]);

  for (long long i = 0; i < lenQ; i++) {
    long long m = Dims[i];
    //MKL_Zomatcopy('C', 'C', m, m, std::complex<double>(1., 0.), Q[i], m, reinterpret_cast<std::complex<double>*>(&hostU[i * block]), bdim);
    omatcopy('C', 'C', m, m, Q[i], m, &hostU[i * block], bdim);
  }
  hipMemcpy(Udata, hostU, block * lenQ * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Vdata, Udata, block * lenQ * sizeof(float), hipMemcpyDeviceToDevice);
  
  hipHostFree(hostU);
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(float));
  //std::fill(hostA, &hostA[block * lenA], make_hipDoubleComplex(0., 0.));
  fill_zero(hostA, &hostA[block * lenA]);

  std::vector<std::tuple<long long, long long, long long>> coo_list(lenA);
  for (long long y = 0; y < M; y++) {
    long long begin = ARows[y];
    long long end = ARows[y + 1];
    std::transform(&ACols[begin], &ACols[end], coo_list.begin() + begin, 
      [&](const long long& x) { return std::make_tuple(y + D, x, std::distance(ACols, &x)); });
  }

  for (long long i = 0; i < lenA; i++) {
    long long y = std::get<0>(coo_list[i]);
    long long x = std::get<1>(coo_list[i]);
    long long M = Dims[y], N = Dims[x];
    //MKL_Zomatcopy('C', 'N', M, N, std::complex<double>(1., 0.), A[i], M, reinterpret_cast<std::complex<double>*>(&hostA[i * block]), bdim);
    omatcopy('C', 'N', M, N, A[i], M, &hostA[i * block], bdim);
  }

  hipMemcpy(Adata, hostA, block * lenA * sizeof(float), hipMemcpyHostToDevice);
  hipHostFree(hostA);

  std::stable_partition(coo_list.begin(), coo_list.end(), 
    [](std::tuple<int64_t, int64_t, int64_t> i) { return std::get<0>(i) == std::get<1>(i); });

  float** hostAptrs, **hostUptrs, **hostVptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostAptrs), lenA * sizeof(float*));
  hipHostMalloc(reinterpret_cast<void**>(&hostUptrs), lenA * sizeof(float*));
  hipHostMalloc(reinterpret_cast<void**>(&hostVptrs), lenA * sizeof(float*));

  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Udata + std::get<1>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block; });

  hipMemcpy(A_SS, hostAptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(U, hostUptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(V, hostVptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);

  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_SR; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RR; });

  hipMemcpy(A_SR, hostAptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS, hostUptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR, hostVptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);

  std::transform(coo_list.begin(), coo_list.begin() + M, hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block + offset_RS; });
  hipMemcpy(V_R, hostVptrs, M * sizeof(float*), hipMemcpyHostToDevice);

  hipHostFree(hostAptrs);
  hipHostFree(hostUptrs);
  hipHostFree(hostVptrs);
}

// needs explicit specialization due to cuBLAS calls
template <>
void H2Factorize<hipDoubleComplex>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  hipDoubleComplex one = make_hipDoubleComplex(1., 0.), zero = make_hipDoubleComplex(0., 0.), minus_one = make_hipDoubleComplex(-1., 0.);
  int info_host = 0;

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);

  hipblasZgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize<hipComplex>::compute(const hipblasComputeType_t COMP) {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  hipComplex one = make_hipComplex(1., 0.), zero = make_hipComplex(0., 0.), minus_one = make_hipComplex(-1., 0.);
  int info_host = 0;

  const auto ALGO = HIPBLAS_GEMM_DEFAULT;

  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_C_32F, N, reinterpret_cast<void**>(A_SS), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_C_32F, N, D, COMP, ALGO);
  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_C_32F, N, reinterpret_cast<void**>(B), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(A_SS), HIP_C_32F, N, D, COMP, ALGO);

  hipblasCgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasCgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasCgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, reinterpret_cast<void*>(&minus_one), reinterpret_cast<void**>(A_SR), HIP_C_32F, N, reinterpret_cast<void**>(A_RS), HIP_C_32F, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(A_SS), HIP_C_32F, N, D, COMP, ALGO);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&U[i]), HIP_R_32F, N, reinterpret_cast<void**>(&A_SS[i]), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_C_32F, N, len, COMP, ALGO);
    hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&V[i]), HIP_R_32F, N, reinterpret_cast<void**>(B), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(&A_SS[i]), HIP_C_32F, N, len, COMP, ALGO);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize<hipComplex>::compute() {
  compute(HIPBLAS_COMPUTE_32F);
}

template <>
void H2Factorize<double>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  double one = 1, zero = 0, minus_one = -1;
  int info_host = 0;

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);

  hipblasDgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasDgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasDgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize<float>::compute(const hipblasComputeType_t COMP) {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  float one = 1, zero = 0, minus_one = -1;
  int info_host = 0;

  const auto ALGO = HIPBLAS_GEMM_DEFAULT;

  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_R_32F, N, reinterpret_cast<void**>(A_SS), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_R_32F, N, D, COMP, ALGO);
  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_R_32F, N, reinterpret_cast<void**>(B), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(A_SS), HIP_R_32F, N, D, COMP, ALGO);

  hipblasSgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasSgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasSgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, reinterpret_cast<void*>(&minus_one), reinterpret_cast<void**>(A_SR), HIP_R_32F, N, reinterpret_cast<void**>(A_RS), HIP_R_32F, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(A_SS), HIP_R_32F, N, D, COMP, ALGO);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&U[i]), HIP_R_32F, N, reinterpret_cast<void**>(&A_SS[i]), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_R_32F, N, len, COMP, ALGO);
    hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&V[i]), HIP_R_32F, N, reinterpret_cast<void**>(B), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(&A_SS[i]), HIP_R_32F, N, len, COMP, ALGO);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize<float>::compute() {
  compute(HIPBLAS_COMPUTE_32F);
}

template <typename DT> template <typename OT>
void H2Factorize<DT>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<OT>& A, int* ipvts) {
  long long block = bdim * bdim;
  long long lenR = bdim - rank;

  DT* hostA;
  int* hostI;
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(DT));
  hipHostMalloc(reinterpret_cast<void**>(&hostI), lenR * M * sizeof(int));
  hipMemcpy(hostA, Adata, block * lenA * sizeof(DT), hipMemcpyDeviceToHost);
  hipMemcpy(hostI, ipiv, lenR * M * sizeof(int), hipMemcpyDeviceToHost);

  std::vector<long long> ipiv_offsets(M);
  std::exclusive_scan(&Dims[D], &Dims[D + M], ipiv_offsets.begin(), 0ll);

  for (long long i = 0; i < M; i++) {
    long long m = Dims[i + D];
    
    std::vector<int> rows(lenR);
    std::iota(rows.begin(), rows.end(), 0);
    for (long long j = lenR - 1; j >= 0; j--) {
      int p = hostI[i * lenR + j] - 1;
      if (p != j)
        std::iter_swap(rows.begin() + j, rows.begin() + p);
    }
    std::copy(rows.begin(), rows.begin() + m, &ipvts[ipiv_offsets[i]]);
    
    // used when not using eigen
    //std::copy(&hostI[i * lenR], &hostI[(i + 1) * lenR], &ipvts[ipiv_offsets[i]]);

    for (long long ij = ARows[i]; ij < ARows[i + 1]; ij++) {
      long long j = ACols[ij];
      long long n = Dims[j];
      //MKL_Zomatcopy('C', 'N', m, n, std::complex<double>(1., 0.), reinterpret_cast<std::complex<double>*>(&hostA[ij * block]), bdim, A[ij], m);
      omatcopy('C', 'N', m, n, &hostA[ij * block], bdim, A[ij], m);
    }
  }

  hipHostFree(hostA);
  hipHostFree(hostI);
}

/* half precision specializations */
H2Factorize<__half>::H2Factorize(long long LD, long long lenA, long long lenQ, hipStream_t stream) : maxA(lenA), maxQ(lenQ), bdim(LD), stream(stream) {
  cublasH = nullptr;
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  long long bsize = LD * LD * sizeof(__half);
  hipMalloc(reinterpret_cast<void**>(&Adata), bsize * lenA);
  hipMalloc(reinterpret_cast<void**>(&Bdata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Udata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Vdata), bsize * lenQ);
  bsize = LD * LD * sizeof(float);
  hipMalloc(reinterpret_cast<void**>(&Adata_float), bsize * lenA);
  hipMalloc(reinterpret_cast<void**>(&Vdata_float), bsize * lenQ);

  long long psize = sizeof(__half*);
  hipMalloc(reinterpret_cast<void**>(&A_SS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_SR), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RR), psize * lenA);

  hipMalloc(reinterpret_cast<void**>(&B), psize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&U), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V_R), psize * lenQ);
  psize = sizeof(float*);
  hipMalloc(reinterpret_cast<void**>(&A_RS_float), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RR_float), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V_R_float), psize * lenQ);

  hipMalloc(reinterpret_cast<void**>(&ipiv), LD * lenQ * sizeof(int));
  hipMalloc(reinterpret_cast<void**>(&info), lenQ * sizeof(int));

  __half** hostB;
  hipHostMalloc(reinterpret_cast<void**>(&hostB), psize * lenQ);
  for (long long i = 0; i < lenQ; i++)
    hostB[i] = &Bdata[i * LD * LD];

  hipMemcpy(B, hostB, psize * lenQ, hipMemcpyHostToDevice);
  hipHostFree(hostB);
}

H2Factorize<__half>::~H2Factorize() {
  hipFree(Adata);
  hipFree(Bdata);
  hipFree(Udata);
  hipFree(Vdata);
  hipFree(Adata_float);
  hipFree(Vdata_float);
  hipFree(A_SS);
  hipFree(A_SR);
  hipFree(A_RS);
  hipFree(A_RR);
  hipFree(A_RS_float);
  hipFree(A_RR_float);
  hipFree(B);
  hipFree(U);
  hipFree(V);
  hipFree(V_R);
  hipFree(V_R_float);
  hipFree(ipiv);
  hipFree(info);
}

void H2Factorize<__half>::setData(long long rank, long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], const MatrixDataContainer<float>& A, const MatrixDataContainer<float>& Q) {
  long long block = bdim * bdim;
  lenD = M;
  lenA = std::min(maxA, A.nblocks());
  long long lenQ = std::min(maxQ, Q.nblocks());
  H2Factorize::rank = rank;

  float* hostU_float, *hostA_float;
  __half* hostU, *hostA;
  hipHostMalloc(reinterpret_cast<void**>(&hostU_float), block * lenQ * sizeof(float));
  fill_zero(hostU_float, &hostU_float[block * lenQ]);

  for (long long i = 0; i < lenQ; i++) {
    long long m = Dims[i];
    omatcopy('C', 'C', m, m, Q[i], m, &hostU_float[i * block], bdim);
  }
  hipHostMalloc(reinterpret_cast<void**>(&hostU), block * lenQ * sizeof(__half));
  for (long long i = 0; i < block * lenQ; i++) {
    hostU[i] = 	__float2half(hostU_float[i]); 
  }

  hipMemcpy(Udata, hostU, block * lenQ * sizeof(__half), hipMemcpyHostToDevice);
  hipMemcpy(Vdata, Udata, block * lenQ * sizeof(__half), hipMemcpyDeviceToDevice);
  
  hipHostFree(hostU_float);
  hipHostFree(hostU);

  hipHostMalloc(reinterpret_cast<void**>(&hostA_float), block * lenA * sizeof(float));
  fill_zero(hostA_float, &hostA_float[block * lenA]);

  std::vector<std::tuple<long long, long long, long long>> coo_list(lenA);
  for (long long y = 0; y < M; y++) {
    long long begin = ARows[y];
    long long end = ARows[y + 1];
    std::transform(&ACols[begin], &ACols[end], coo_list.begin() + begin, 
      [&](const long long& x) { return std::make_tuple(y + D, x, std::distance(ACols, &x)); });
  }

  for (long long i = 0; i < lenA; i++) {
    long long y = std::get<0>(coo_list[i]);
    long long x = std::get<1>(coo_list[i]);
    long long M = Dims[y], N = Dims[x];
    omatcopy('C', 'N', M, N, A[i], M, &hostA_float[i * block], bdim);
  }
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(__half));
  for (long long i = 0; i < block * lenA; i++) {
    hostA[i] = 	__float2half(hostA_float[i]); 
  }

  hipMemcpy(Adata, hostA, block * lenA * sizeof(__half), hipMemcpyHostToDevice);
  hipHostFree(hostA_float);
  hipHostFree(hostA);

  std::stable_partition(coo_list.begin(), coo_list.end(), 
    [](std::tuple<int64_t, int64_t, int64_t> i) { return std::get<0>(i) == std::get<1>(i); });

   __half** hostAptrs, **hostUptrs, **hostVptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostAptrs), lenA * sizeof(__half*));
  hipHostMalloc(reinterpret_cast<void**>(&hostUptrs), lenA * sizeof(__half*));
  hipHostMalloc(reinterpret_cast<void**>(&hostVptrs), lenA * sizeof(__half*));

  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Udata + std::get<1>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block; });

  hipMemcpy(A_SS, hostAptrs, lenA * sizeof(__half*), hipMemcpyHostToDevice);
  hipMemcpy(U, hostUptrs, lenA * sizeof(__half*), hipMemcpyHostToDevice);
  hipMemcpy(V, hostVptrs, lenA * sizeof(__half*), hipMemcpyHostToDevice);

  float** hostA_RRptrs, **hostA_RSptrs, **hostV_Rptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostA_RRptrs), lenA * sizeof(float*));
  hipHostMalloc(reinterpret_cast<void**>(&hostA_RSptrs), lenA * sizeof(float*));
  hipHostMalloc(reinterpret_cast<void**>(&hostV_Rptrs), lenA * sizeof(float*));
  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_SR; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RR; });
  std::transform(coo_list.begin(), coo_list.end(), hostA_RSptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata_float + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostA_RRptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata_float + std::get<2>(i) * block + offset_RR; });

  hipMemcpy(A_SR, hostAptrs, lenA * sizeof(__half*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS, hostUptrs, lenA * sizeof(__half*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR, hostVptrs, lenA * sizeof(__half*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS_float, hostA_RSptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR_float, hostA_RRptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);

  std::transform(coo_list.begin(), coo_list.begin() + M, hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.begin() + M, hostV_Rptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata_float + std::get<0>(i) * block + offset_RS; });
  hipMemcpy(V_R, hostVptrs, M * sizeof(__half*), hipMemcpyHostToDevice);
  hipMemcpy(V_R_float, hostV_Rptrs, M * sizeof(__half*), hipMemcpyHostToDevice);

  hipHostFree(hostAptrs);
  hipHostFree(hostUptrs);
  hipHostFree(hostVptrs);
  hipHostFree(hostA_RSptrs);
  hipHostFree(hostA_RRptrs);
  hipHostFree(hostV_Rptrs);
}

void H2Factorize<__half>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  __half one = 1, zero = 0, minus_one = -1;
  int info_host = 0;

  hipblasHgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasHgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);
  tofloat<<<(N * N * lenA + 1023) / 1024, 1024, 0, stream>>>(Adata, Adata_float, N * N * lenA);
  
  hipblasSgetrfBatched(cublasH, R, A_RR_float, N, ipiv, info, D);
  hipblasSgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR_float, N, ipiv, A_RS_float, N, &info_host, D);
  hipblasSgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR_float, N, ipiv, V_R_float, N, &info_host, D);
  tohalf<<<(N * N * lenA + 1023) / 1024, 1024, 0, stream>>>(Adata_float, Adata, N * N * lenA);
  tohalf<<<(N * N * maxQ + 1023) / 1024, 1024, 0, stream>>>(Adata_float, Adata, N * N * maxQ);
  //tohalf<<<maxA, bdim * bdim>>>(Adata_float, Adata);
  //tohalf<<<maxQ, bdim * bdim>>>(Vdata_float, Vdata);

  hipblasHgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasHgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasHgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }
  //tofloat<<<(N * N * lenA + 1023) / 1024, 1024, 0, stream>>>(Adata, Adata_float, N * N * lenA);
  //tofloat<<<maxA, bdim * bdim>>>(Adata, Adata_float);

  hipStreamSynchronize(stream);
}

void H2Factorize<__half>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<float>& A, int* ipvts) {
  long long block = bdim * bdim;
  long long lenR = bdim - rank;

  //hipStreamSynchronize(stream);
  //hipDeviceSynchronize();
  //tofloat<<<(block * lenA + 1023) / 1024, 1024>>>(Adata, Adata_float, block * lenA);
  //tofloat<<<maxA, bdim * bdim>>>(Adata, Adata_float);
  //hipDeviceSynchronize();
  __half* hostA;
  float* hostA_float;
  int* hostI;
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(__half));
  hipHostMalloc(reinterpret_cast<void**>(&hostA_float), block * lenA * sizeof(float));
  hipHostMalloc(reinterpret_cast<void**>(&hostI), lenR * M * sizeof(int));
  hipMemcpy(hostA, Adata, block * lenA * sizeof(__half), hipMemcpyDeviceToHost);
  //hipMemcpy(hostA_float, Adata_float, block * lenA * sizeof(float), hipMemcpyDeviceToHost);
  for (long long i = 0; i < block * lenA; i++) {
    hostA_float[i] = __half2float(hostA[i]); 
  }
  hipMemcpy(hostI, ipiv, lenR * M * sizeof(int), hipMemcpyDeviceToHost);

  std::vector<long long> ipiv_offsets(M);
  std::exclusive_scan(&Dims[D], &Dims[D + M], ipiv_offsets.begin(), 0ll);

  for (long long i = 0; i < M; i++) {
    long long m = Dims[i + D];
    
    std::vector<int> rows(lenR);
    std::iota(rows.begin(), rows.end(), 0);
    for (long long j = lenR - 1; j >= 0; j--) {
      int p = hostI[i * lenR + j] - 1;
      if (p != j)
        std::iter_swap(rows.begin() + j, rows.begin() + p);
    }
    std::copy(rows.begin(), rows.begin() + m, &ipvts[ipiv_offsets[i]]);
    
    // used when not using eigen
    //std::copy(&hostI[i * lenR], &hostI[(i + 1) * lenR], &ipvts[ipiv_offsets[i]]);

    for (long long ij = ARows[i]; ij < ARows[i + 1]; ij++) {
      long long j = ACols[ij];
      long long n = Dims[j];
      omatcopy('C', 'N', m, n, &hostA_float[ij * block], bdim, A[ij], m);
    }
  }

  hipHostFree(hostA);
  hipHostFree(hostI);
}

/* explicit template instantiation */
// complex double
template class H2Factorize<hipDoubleComplex>;
template void H2Factorize<hipDoubleComplex>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<std::complex<double>>&, const MatrixDataContainer<std::complex<double>>&);
template void H2Factorize<hipDoubleComplex>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<std::complex<double>>& A, int* ipvts);
// complex float
template class H2Factorize<hipComplex>;
template void H2Factorize<hipComplex>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<std::complex<float>>&, const MatrixDataContainer<std::complex<float>>&);
template void H2Factorize<hipComplex>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<std::complex<float>>& A, int* ipvts);
// double
template class H2Factorize<double>;
template void H2Factorize<double>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<double>&, const MatrixDataContainer<double>&);
template void H2Factorize<double>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<double>& A, int* ipvts);
// float
template class H2Factorize<float>;
template void H2Factorize<float>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<float>&, const MatrixDataContainer<float>&);
template void H2Factorize<float>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<float>& A, int* ipvts);


template <typename DT>
H2Factorize2<DT>::H2Factorize2(long long LD, long long lenA, long long lenQ, hipStream_t stream) : maxA(lenA), maxQ(lenQ), bdim(LD), stream(stream) {
  cublasH = nullptr;
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  long long bsize = LD * LD * sizeof(DT);
  hipMalloc(reinterpret_cast<void**>(&Adata), bsize * lenA);
  hipMalloc(reinterpret_cast<void**>(&Bdata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Udata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Vdata), bsize * lenQ);

  long long psize = sizeof(DT*);
  hipMalloc(reinterpret_cast<void**>(&A_SS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_SR), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RR), psize * lenA);

  hipMalloc(reinterpret_cast<void**>(&B), psize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&U), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V_R), psize * lenQ);

  hipMalloc(reinterpret_cast<void**>(&ipiv), LD * lenQ * sizeof(int));
  hipMalloc(reinterpret_cast<void**>(&info), lenQ * sizeof(int));

  DT** hostB;
  hipHostMalloc(reinterpret_cast<void**>(&hostB), psize * lenQ);
  for (long long i = 0; i < lenQ; i++)
    hostB[i] = &Bdata[i * LD * LD];

  hipMemcpy(B, hostB, psize * lenQ, hipMemcpyHostToDevice);
  Bptr = std::vector<DT*>(lenQ);
  std::memcpy(Bptr.data(), hostB, psize * lenQ);
  hipHostFree(hostB);
}

template <typename DT>
H2Factorize2<DT>::~H2Factorize2() {
  hipFree(Adata);
  hipFree(Bdata);
  hipFree(Udata);
  hipFree(Vdata);
  hipFree(A_SS);
  hipFree(A_SR);
  hipFree(A_RS);
  hipFree(A_RR);
  hipFree(B);
  hipFree(U);
  hipFree(V);
  hipFree(V_R);
  hipFree(ipiv);
  hipFree(info);
}

template <typename DT> template <typename OT>
void H2Factorize2<DT>::setData(long long rank, long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], const MatrixDataContainer<OT>& A, const MatrixDataContainer<OT>& Q) {
  long long block = bdim * bdim;
  lenD = M;
  lenA = std::min(maxA, A.nblocks());
  long long lenQ = std::min(maxQ, Q.nblocks());
  H2Factorize2::rank = rank;

  DT* hostU, *hostA;
  hipHostMalloc(reinterpret_cast<void**>(&hostU), block * lenQ * sizeof(DT));
  //std::fill(hostU, &hostU[block * lenQ], make_hipDoubleComplex(0., 0.));
  fill_zero(hostU, &hostU[block * lenQ]);

  for (long long i = 0; i < lenQ; i++) {
    long long m = Dims[i];
    //MKL_Zomatcopy('C', 'C', m, m, std::complex<double>(1., 0.), Q[i], m, reinterpret_cast<std::complex<double>*>(&hostU[i * block]), bdim);
    omatcopy('C', 'C', m, m, Q[i], m, &hostU[i * block], bdim);
  }
  hipMemcpy(Udata, hostU, block * lenQ * sizeof(DT), hipMemcpyHostToDevice);
  hipMemcpy(Vdata, Udata, block * lenQ * sizeof(DT), hipMemcpyDeviceToDevice);
  
  hipHostFree(hostU);
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(DT));
  //std::fill(hostA, &hostA[block * lenA], make_hipDoubleComplex(0., 0.));
  fill_zero(hostA, &hostA[block * lenA]);

  std::vector<std::tuple<long long, long long, long long>> coo_list(lenA);
  for (long long y = 0; y < M; y++) {
    long long begin = ARows[y];
    long long end = ARows[y + 1];
    std::transform(&ACols[begin], &ACols[end], coo_list.begin() + begin, 
      [&](const long long& x) { return std::make_tuple(y + D, x, std::distance(ACols, &x)); });
  }

  for (long long i = 0; i < lenA; i++) {
    long long y = std::get<0>(coo_list[i]);
    long long x = std::get<1>(coo_list[i]);
    long long M = Dims[y], N = Dims[x];
    //MKL_Zomatcopy('C', 'N', M, N, std::complex<double>(1., 0.), A[i], M, reinterpret_cast<std::complex<double>*>(&hostA[i * block]), bdim);
    omatcopy('C', 'N', M, N, A[i], M, &hostA[i * block], bdim);
  }

  hipMemcpy(Adata, hostA, block * lenA * sizeof(DT), hipMemcpyHostToDevice);
  hipHostFree(hostA);

  std::stable_partition(coo_list.begin(), coo_list.end(), 
    [](std::tuple<int64_t, int64_t, int64_t> i) { return std::get<0>(i) == std::get<1>(i); });

  DT** hostAptrs, **hostUptrs, **hostVptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostAptrs), lenA * sizeof(DT*));
  hipHostMalloc(reinterpret_cast<void**>(&hostUptrs), lenA * sizeof(DT*));
  hipHostMalloc(reinterpret_cast<void**>(&hostVptrs), lenA * sizeof(DT*));

  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Udata + std::get<1>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block; });

  hipMemcpy(A_SS, hostAptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(U, hostUptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(V, hostVptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);

  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_SR; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RR; });

  hipMemcpy(A_SR, hostAptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS, hostUptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR, hostVptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);

  std::transform(coo_list.begin(), coo_list.begin() + M, hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block + offset_RS; });
  hipMemcpy(V_R, hostVptrs, M * sizeof(DT*), hipMemcpyHostToDevice);

  hipHostFree(hostAptrs);
  hipHostFree(hostUptrs);
  hipHostFree(hostVptrs);
}

// specialization for experimental purposes
template <> template<>
void H2Factorize2<float>::setData(long long rank, long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], const MatrixDataContainer<float>& A, const MatrixDataContainer<float>& Q) {
  long long block = bdim * bdim;
  lenD = M;
  lenA = std::min(maxA, A.nblocks());
  long long lenQ = std::min(maxQ, Q.nblocks());
  H2Factorize2::rank = rank;

  float* hostU, *hostA;
  hipHostMalloc(reinterpret_cast<void**>(&hostU), block * lenQ * sizeof(float));
  //std::fill(hostU, &hostU[block * lenQ], make_hipDoubleComplex(0., 0.));
  fill_zero(hostU, &hostU[block * lenQ]);

  for (long long i = 0; i < lenQ; i++) {
    long long m = Dims[i];
    //MKL_Zomatcopy('C', 'C', m, m, std::complex<double>(1., 0.), Q[i], m, reinterpret_cast<std::complex<double>*>(&hostU[i * block]), bdim);
    omatcopy('C', 'C', m, m, Q[i], m, &hostU[i * block], bdim);
  }
  hipMemcpy(Udata, hostU, block * lenQ * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Vdata, Udata, block * lenQ * sizeof(float), hipMemcpyDeviceToDevice);
  
  hipHostFree(hostU);
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(float));
  //std::fill(hostA, &hostA[block * lenA], make_hipDoubleComplex(0., 0.));
  fill_zero(hostA, &hostA[block * lenA]);

  std::vector<std::tuple<long long, long long, long long>> coo_list(lenA);
  for (long long y = 0; y < M; y++) {
    long long begin = ARows[y];
    long long end = ARows[y + 1];
    std::transform(&ACols[begin], &ACols[end], coo_list.begin() + begin, 
      [&](const long long& x) { return std::make_tuple(y + D, x, std::distance(ACols, &x)); });
  }

  for (long long i = 0; i < lenA; i++) {
    long long y = std::get<0>(coo_list[i]);
    long long x = std::get<1>(coo_list[i]);
    long long M = Dims[y], N = Dims[x];
    //MKL_Zomatcopy('C', 'N', M, N, std::complex<double>(1., 0.), A[i], M, reinterpret_cast<std::complex<double>*>(&hostA[i * block]), bdim);
    omatcopy('C', 'N', M, N, A[i], M, &hostA[i * block], bdim);
  }

  hipMemcpy(Adata, hostA, block * lenA * sizeof(float), hipMemcpyHostToDevice);
  hipHostFree(hostA);

  std::stable_partition(coo_list.begin(), coo_list.end(), 
    [](std::tuple<int64_t, int64_t, int64_t> i) { return std::get<0>(i) == std::get<1>(i); });

  float** hostAptrs, **hostUptrs, **hostVptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostAptrs), lenA * sizeof(float*));
  hipHostMalloc(reinterpret_cast<void**>(&hostUptrs), lenA * sizeof(float*));
  hipHostMalloc(reinterpret_cast<void**>(&hostVptrs), lenA * sizeof(float*));

  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Udata + std::get<1>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block; });

  hipMemcpy(A_SS, hostAptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(U, hostUptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(V, hostVptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  Aptr_SS = std::vector<float*>(lenA);
  Uptr = std::vector<float*>(lenA);
  Vptr = std::vector<float*>(lenA);
  std::memcpy(Aptr_SS.data(), hostAptrs, lenA * sizeof(float*));
  std::memcpy(Uptr.data(), hostUptrs, lenA * sizeof(float*));
  std::memcpy(Vptr.data(), hostVptrs, lenA * sizeof(float*));


  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_SR; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RR; });

  hipMemcpy(A_SR, hostAptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS, hostUptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR, hostVptrs, lenA * sizeof(float*), hipMemcpyHostToDevice);
  Aptr_RS = std::vector<float*>(lenA);
  Aptr_SR = std::vector<float*>(lenA);
  std::memcpy(Aptr_SR.data(), hostAptrs, lenA * sizeof(float*));
  std::memcpy(Aptr_RS.data(), hostUptrs, lenA * sizeof(float*));

  std::transform(coo_list.begin(), coo_list.begin() + M, hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block + offset_RS; });
  hipMemcpy(V_R, hostVptrs, M * sizeof(float*), hipMemcpyHostToDevice);

  hipHostFree(hostAptrs);
  hipHostFree(hostUptrs);
  hipHostFree(hostVptrs);
}

// needs explicit specialization due to cuBLAS calls
template <>
void H2Factorize2<hipDoubleComplex>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  hipDoubleComplex one = make_hipDoubleComplex(1., 0.), zero = make_hipDoubleComplex(0., 0.), minus_one = make_hipDoubleComplex(-1., 0.);
  int info_host = 0;

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);

  hipblasZgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize2<hipComplex>::compute(const hipblasComputeType_t COMP) {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  hipComplex one = make_hipComplex(1., 0.), zero = make_hipComplex(0., 0.), minus_one = make_hipComplex(-1., 0.);
  int info_host = 0;

  const auto ALGO = HIPBLAS_GEMM_DEFAULT;

  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_C_32F, N, reinterpret_cast<void**>(A_SS), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_C_32F, N, D, COMP, ALGO);
  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_C_32F, N, reinterpret_cast<void**>(B), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(A_SS), HIP_C_32F, N, D, COMP, ALGO);

  hipblasCgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasCgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasCgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, reinterpret_cast<void*>(&minus_one), reinterpret_cast<void**>(A_SR), HIP_C_32F, N, reinterpret_cast<void**>(A_RS), HIP_C_32F, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(A_SS), HIP_C_32F, N, D, COMP, ALGO);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&U[i]), HIP_R_32F, N, reinterpret_cast<void**>(&A_SS[i]), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_C_32F, N, len, COMP, ALGO);
    hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&V[i]), HIP_R_32F, N, reinterpret_cast<void**>(B), HIP_C_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(&A_SS[i]), HIP_C_32F, N, len, COMP, ALGO);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize2<hipComplex>::compute() {
  compute(HIPBLAS_COMPUTE_32F);
}

template <>
void H2Factorize2<double>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  double one = 1, zero = 0, minus_one = -1;
  int info_host = 0;

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);

  hipblasDgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasDgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasDgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasDgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }

  hipStreamSynchronize(stream);
}

template <>
void H2Factorize2<float>::compute(const hipblasComputeType_t COMP) {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  float one = 1, zero = 0, minus_one = -1;
  int info_host = 0;

  const auto ALGO = HIPBLAS_GEMM_DEFAULT;
  for (long long i=0; i<D; ++i) {
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, Uptr[i], N, Aptr_SS[i], N, &zero, Bptr[i], N);
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, Uptr[i], N, Bptr[i], N, &zero, Aptr_SS[i], N);
  }
  //hipblasSgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[0], N, &A_SS[0], N, &zero, &B[0], N, 1);
  //hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_R_32F, N, reinterpret_cast<void**>(A_SS), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_R_32F, N, D, COMP, ALGO);
  //hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(U), HIP_R_32F, N, reinterpret_cast<void**>(B), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(A_SS), HIP_R_32F, N, D, COMP, ALGO);

  hipblasSgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasSgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasSgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  for (long long i=0; i<D; ++i) {
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, Aptr_SR[i], N, Aptr_RS[i], N, &one, Aptr_SS[i], N);
  }
  //hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, reinterpret_cast<void*>(&minus_one), reinterpret_cast<void**>(A_SR), HIP_R_32F, N, reinterpret_cast<void**>(A_RS), HIP_R_32F, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(A_SS), HIP_R_32F, N, D, COMP, ALGO);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    for (long long j=0; j<len; ++j) {
      // TODO note that this loop does not deliver the exact same result as the batched version
      hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, Uptr[i+j], N, Aptr_SS[i+j], N, &zero, Bptr[j], N);
      hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, Vptr[i+j], N, Bptr[j], N, &zero, Aptr_SS[i+j], N);
    }
    //hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&U[i]), HIP_R_32F, N, reinterpret_cast<void**>(&A_SS[i]), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(B), HIP_R_32F, N, len, COMP, ALGO);
    //hipblasGemmBatchedEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, reinterpret_cast<void*>(&one), reinterpret_cast<void**>(&V[i]), HIP_R_32F, N, reinterpret_cast<void**>(B), HIP_R_32F, N, reinterpret_cast<void*>(&zero), reinterpret_cast<void**>(&A_SS[i]), HIP_R_32F, N, len, COMP, ALGO);
  }
  
  hipStreamSynchronize(stream);
}

template <>
void H2Factorize2<float>::compute() {
  compute(HIPBLAS_COMPUTE_32F);
}

template <typename DT> template <typename OT>
void H2Factorize2<DT>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<OT>& A, int* ipvts) {
  long long block = bdim * bdim;
  long long lenR = bdim - rank;

  DT* hostA;
  int* hostI;
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(DT));
  hipHostMalloc(reinterpret_cast<void**>(&hostI), lenR * M * sizeof(int));
  hipMemcpy(hostA, Adata, block * lenA * sizeof(DT), hipMemcpyDeviceToHost);
  hipMemcpy(hostI, ipiv, lenR * M * sizeof(int), hipMemcpyDeviceToHost);

  std::vector<long long> ipiv_offsets(M);
  std::exclusive_scan(&Dims[D], &Dims[D + M], ipiv_offsets.begin(), 0ll);

  for (long long i = 0; i < M; i++) {
    long long m = Dims[i + D];
    
    std::vector<int> rows(lenR);
    std::iota(rows.begin(), rows.end(), 0);
    for (long long j = lenR - 1; j >= 0; j--) {
      int p = hostI[i * lenR + j] - 1;
      if (p != j)
        std::iter_swap(rows.begin() + j, rows.begin() + p);
    }
    std::copy(rows.begin(), rows.begin() + m, &ipvts[ipiv_offsets[i]]);
    
    // used when not using eigen
    //std::copy(&hostI[i * lenR], &hostI[(i + 1) * lenR], &ipvts[ipiv_offsets[i]]);

    for (long long ij = ARows[i]; ij < ARows[i + 1]; ij++) {
      long long j = ACols[ij];
      long long n = Dims[j];
      //MKL_Zomatcopy('C', 'N', m, n, std::complex<double>(1., 0.), reinterpret_cast<std::complex<double>*>(&hostA[ij * block]), bdim, A[ij], m);
      omatcopy('C', 'N', m, n, &hostA[ij * block], bdim, A[ij], m);
    }
  }

  hipHostFree(hostA);
  hipHostFree(hostI);
}

/* explicit template instantiation */
// complex double
template class H2Factorize2<hipDoubleComplex>;
template void H2Factorize2<hipDoubleComplex>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<std::complex<double>>&, const MatrixDataContainer<std::complex<double>>&);
template void H2Factorize2<hipDoubleComplex>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<std::complex<double>>& A, int* ipvts);
// complex float
template class H2Factorize2<hipComplex>;
template void H2Factorize2<hipComplex>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<std::complex<float>>&, const MatrixDataContainer<std::complex<float>>&);
template void H2Factorize2<hipComplex>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<std::complex<float>>& A, int* ipvts);
// double
template class H2Factorize2<double>;
template void H2Factorize2<double>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<double>&, const MatrixDataContainer<double>&);
template void H2Factorize2<double>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<double>& A, int* ipvts);
// float
template class H2Factorize2<float>;
template void H2Factorize2<float>::setData(long long, long long, long long, const long long[], const long long [], const long long[], const MatrixDataContainer<float>&, const MatrixDataContainer<float>&);
template void H2Factorize2<float>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<float>& A, int* ipvts);