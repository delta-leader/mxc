#include "hip/hip_runtime.h"

#include <factorize.cuh>
#include <algorithm>
#include <numeric>
#include <tuple>
#include <mkl.h>

inline void fill_zero(hipDoubleComplex* start, hipDoubleComplex* end) {
  std::fill(start, end, make_hipDoubleComplex(0., 0.));
}

void omatcopy(char ordering, char trans, size_t rows, size_t cols, const std::complex<double> *SRC, size_t src_stride, hipDoubleComplex *DST, size_t dst_stride) {
  MKL_Zomatcopy(ordering, trans, rows, cols, std::complex<double>(1., 0.), SRC, src_stride, reinterpret_cast<std::complex<double>*>(DST), dst_stride);
}
void omatcopy(char ordering, char trans, size_t rows, size_t cols, hipDoubleComplex *SRC, size_t src_stride, std::complex<double> *DST, size_t dst_stride) {
  MKL_Zomatcopy(ordering, trans, rows, cols, std::complex<double>(1., 0.), reinterpret_cast<std::complex<double>*>(SRC), src_stride, DST, dst_stride);
}

template <typename DT>
H2Factorize<DT>::H2Factorize(long long LD, long long lenA, long long lenQ, hipStream_t stream) : maxA(lenA), maxQ(lenQ), bdim(LD), stream(stream) {
  cublasH = nullptr;
  hipblasCreate(&cublasH);
  hipblasSetStream(cublasH, stream);

  long long bsize = LD * LD * sizeof(DT);
  hipMalloc(reinterpret_cast<void**>(&Adata), bsize * lenA);
  hipMalloc(reinterpret_cast<void**>(&Bdata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Udata), bsize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&Vdata), bsize * lenQ);

  long long psize = sizeof(DT*);
  hipMalloc(reinterpret_cast<void**>(&A_SS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_SR), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RS), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&A_RR), psize * lenA);

  hipMalloc(reinterpret_cast<void**>(&B), psize * lenQ);
  hipMalloc(reinterpret_cast<void**>(&U), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V), psize * lenA);
  hipMalloc(reinterpret_cast<void**>(&V_R), psize * lenQ);

  hipMalloc(reinterpret_cast<void**>(&ipiv), LD * lenQ * sizeof(int));
  hipMalloc(reinterpret_cast<void**>(&info), lenQ * sizeof(int));

  DT** hostB;
  hipHostMalloc(reinterpret_cast<void**>(&hostB), psize * lenQ);
  for (long long i = 0; i < lenQ; i++)
    hostB[i] = &Bdata[i * LD * LD];

  hipMemcpy(B, hostB, psize * lenQ, hipMemcpyHostToDevice);
  hipHostFree(hostB);
}

template <typename DT>
H2Factorize<DT>::~H2Factorize() {
  hipFree(Adata);
  hipFree(Bdata);
  hipFree(Udata);
  hipFree(Vdata);
  hipFree(A_SS);
  hipFree(A_SR);
  hipFree(A_RS);
  hipFree(A_RR);
  hipFree(B);
  hipFree(U);
  hipFree(V);
  hipFree(V_R);
  hipFree(ipiv);
  hipFree(info);
}

template <typename DT>
void H2Factorize<DT>::setData(long long rank, long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], const MatrixDataContainer<std::complex<double>>& A, const MatrixDataContainer<std::complex<double>>& Q) {
  long long block = bdim * bdim;
  lenD = M;
  lenA = std::min(maxA, A.nblocks());
  long long lenQ = std::min(maxQ, Q.nblocks());
  H2Factorize::rank = rank;

  DT* hostU, *hostA;
  hipHostMalloc(reinterpret_cast<void**>(&hostU), block * lenQ * sizeof(DT));
  //std::fill(hostU, &hostU[block * lenQ], make_hipDoubleComplex(0., 0.));
  fill_zero(hostU, &hostU[block * lenQ]);

  for (long long i = 0; i < lenQ; i++) {
    long long m = Dims[i];
    //MKL_Zomatcopy('C', 'C', m, m, std::complex<double>(1., 0.), Q[i], m, reinterpret_cast<std::complex<double>*>(&hostU[i * block]), bdim);
    omatcopy('C', 'C', m, m, Q[i], m, &hostU[i * block], bdim);
  }
  hipMemcpy(Udata, hostU, block * lenQ * sizeof(DT), hipMemcpyHostToDevice);
  hipMemcpy(Vdata, Udata, block * lenQ * sizeof(DT), hipMemcpyDeviceToDevice);
  
  hipHostFree(hostU);
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(DT));
  //std::fill(hostA, &hostA[block * lenA], make_hipDoubleComplex(0., 0.));
  fill_zero(hostA, &hostA[block * lenA]);

  std::vector<std::tuple<long long, long long, long long>> coo_list(lenA);
  for (long long y = 0; y < M; y++) {
    long long begin = ARows[y];
    long long end = ARows[y + 1];
    std::transform(&ACols[begin], &ACols[end], coo_list.begin() + begin, 
      [&](const long long& x) { return std::make_tuple(y + D, x, std::distance(ACols, &x)); });
  }

  for (long long i = 0; i < lenA; i++) {
    long long y = std::get<0>(coo_list[i]);
    long long x = std::get<1>(coo_list[i]);
    long long M = Dims[y], N = Dims[x];
    //MKL_Zomatcopy('C', 'N', M, N, std::complex<double>(1., 0.), A[i], M, reinterpret_cast<std::complex<double>*>(&hostA[i * block]), bdim);
    omatcopy('C', 'N', M, N, A[i], M, &hostA[i * block], bdim);
  }

  hipMemcpy(Adata, hostA, block * lenA * sizeof(DT), hipMemcpyHostToDevice);
  hipHostFree(hostA);

  std::stable_partition(coo_list.begin(), coo_list.end(), 
    [](std::tuple<int64_t, int64_t, int64_t> i) { return std::get<0>(i) == std::get<1>(i); });

  DT** hostAptrs, **hostUptrs, **hostVptrs;
  hipHostMalloc(reinterpret_cast<void**>(&hostAptrs), lenA * sizeof(DT*));
  hipHostMalloc(reinterpret_cast<void**>(&hostUptrs), lenA * sizeof(DT*));
  hipHostMalloc(reinterpret_cast<void**>(&hostVptrs), lenA * sizeof(DT*));

  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Udata + std::get<1>(i) * block; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block; });

  hipMemcpy(A_SS, hostAptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(U, hostUptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(V, hostVptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);

  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  std::transform(coo_list.begin(), coo_list.end(), hostAptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_SR; });
  std::transform(coo_list.begin(), coo_list.end(), hostUptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RS; });
  std::transform(coo_list.begin(), coo_list.end(), hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Adata + std::get<2>(i) * block + offset_RR; });

  hipMemcpy(A_SR, hostAptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(A_RS, hostUptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);
  hipMemcpy(A_RR, hostVptrs, lenA * sizeof(DT*), hipMemcpyHostToDevice);

  std::transform(coo_list.begin(), coo_list.begin() + M, hostVptrs, [=](std::tuple<int64_t, int64_t, int64_t> i) { return Vdata + std::get<0>(i) * block + offset_RS; });
  hipMemcpy(V_R, hostVptrs, M * sizeof(DT*), hipMemcpyHostToDevice);

  hipHostFree(hostAptrs);
  hipHostFree(hostUptrs);
  hipHostFree(hostVptrs);
}

template <typename DT>
void H2Factorize<DT>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  DT one = make_hipDoubleComplex(1., 0.), zero = make_hipDoubleComplex(0., 0.), minus_one = make_hipDoubleComplex(-1., 0.);
  int info_host = 0;

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);

  hipblasZgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }

  hipStreamSynchronize(stream);
}

// needs explicit specialization due to cuBLAS calls
template <>
void H2Factorize<hipDoubleComplex>::compute() {
  long long N = bdim, S = rank, R = N - S;
  long long D = lenD;
  hipDoubleComplex one = make_hipDoubleComplex(1., 0.), zero = make_hipDoubleComplex(0., 0.), minus_one = make_hipDoubleComplex(-1., 0.);
  int info_host = 0;

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, A_SS, N, &zero, B, N, D);
  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, U, N, B, N, &zero, A_SS, N, D);

  hipblasZgetrfBatched(cublasH, R, A_RR, N, ipiv, info, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, S, A_RR, N, ipiv, A_RS, N, &info_host, D);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, R, N, A_RR, N, ipiv, V_R, N, &info_host, D);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, S, S, R, &minus_one, A_SR, N, A_RS, N, &one, A_SS, N, D);

  for (int64_t i = D; i < lenA; i += maxQ) {
    int64_t len = std::min(lenA - i, maxQ);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &U[i], N, &A_SS[i], N, &zero, B, N, len);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, &one, &V[i], N, B, N, &zero, &A_SS[i], N, len);
  }

  hipStreamSynchronize(stream);
}

template <typename DT>
void H2Factorize<DT>::getResults(long long D, long long M, const long long ARows[], const long long ACols[], const long long Dims[], MatrixDataContainer<std::complex<double>>& A, int* ipvts) {
  long long block = bdim * bdim;
  long long lenR = bdim - rank;

  DT* hostA;
  int* hostI;
  hipHostMalloc(reinterpret_cast<void**>(&hostA), block * lenA * sizeof(DT));
  hipHostMalloc(reinterpret_cast<void**>(&hostI), lenR * M * sizeof(int));
  hipMemcpy(hostA, Adata, block * lenA * sizeof(DT), hipMemcpyDeviceToHost);
  hipMemcpy(hostI, ipiv, lenR * M * sizeof(int), hipMemcpyDeviceToHost);

  std::vector<long long> ipiv_offsets(M);
  std::exclusive_scan(&Dims[D], &Dims[D + M], ipiv_offsets.begin(), 0ll);

  for (long long i = 0; i < M; i++) {
    long long m = Dims[i + D];

    std::vector<int> rows(lenR);
    std::iota(rows.begin(), rows.end(), 0);
    for (long long j = lenR - 1; j >= 0; j--) {
      int p = hostI[i * lenR + j] - 1;
      if (p != j)
        std::iter_swap(rows.begin() + j, rows.begin() + p);
    }
    std::copy(rows.begin(), rows.begin() + m, &ipvts[ipiv_offsets[i]]);

    // used when not using eigen
    //std::copy(&hostI[i * lenR], &hostI[(i + 1) * lenR], &ipvts[ipiv_offsets[i]]);

    for (long long ij = ARows[i]; ij < ARows[i + 1]; ij++) {
      long long j = ACols[ij];
      long long n = Dims[j];
      MKL_Zomatcopy('C', 'N', m, n, std::complex<double>(1., 0.), reinterpret_cast<std::complex<double>*>(&hostA[ij * block]), bdim, A[ij], m);
      omatcopy('C', 'N', m, n, &hostA[ij * block], bdim, A[ij], m);
    }
  }

  hipHostFree(hostA);
  hipHostFree(hostI);
}

/* explicit template instantiation */
// complex double
template class H2Factorize<hipDoubleComplex>;
