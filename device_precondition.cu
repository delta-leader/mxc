#include "hip/hip_runtime.h"

#include <device_factorize.cuh>
#include <comm-mpi.hpp>

#include <numeric>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/transform.h>
#include <thrust/gather.h>
#include <thrust/partition.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/inner_product.h>

#include <iostream>

/* explicit template instantiation */
// complex double
template void createMatrixDesc(deviceMatrixDesc_t<std::complex<double>>* desc, long long bdim, long long rank, deviceMatrixDesc_t<std::complex<double>> lower, H2Matrix<std::complex<double>>& matrix); 
template void destroyMatrixDesc(deviceMatrixDesc_t<std::complex<double>> desc);
template void copyDataInMatrixDesc(deviceMatrixDesc_t<std::complex<double>> desc, const std::complex<double>* A, const std::complex<double>* U, hipStream_t stream);
template void copyDataOutMatrixDesc(deviceMatrixDesc_t<std::complex<double>> desc, std::complex<double>* A, std::complex<double>* V, hipStream_t stream);
template int check_info(deviceMatrixDesc_t<std::complex<double>> A, const long long);
// complex float
template void createMatrixDesc(deviceMatrixDesc_t<std::complex<float>>* desc, long long bdim, long long rank, deviceMatrixDesc_t<std::complex<float>> lower,  H2Matrix<std::complex<float>>& matrix); 
template void destroyMatrixDesc(deviceMatrixDesc_t<std::complex<float>> desc);
template void copyDataInMatrixDesc(deviceMatrixDesc_t<std::complex<float>> desc, const std::complex<float>* A, const std::complex<float>* U, hipStream_t stream);
template void copyDataOutMatrixDesc(deviceMatrixDesc_t<std::complex<float>> desc, std::complex<float>* A, std::complex<float>* V, hipStream_t stream);
template int check_info(deviceMatrixDesc_t<std::complex<float>> A, const long long);
// double
template void createMatrixDesc(deviceMatrixDesc_t<double>* desc, long long bdim, long long rank, deviceMatrixDesc_t<double> lower, H2Matrix<double>& matrix); 
template void destroyMatrixDesc(deviceMatrixDesc_t<double> desc);
template void copyDataInMatrixDesc(deviceMatrixDesc_t<double> desc, const double* A, const double* U, hipStream_t stream);
template void copyDataOutMatrixDesc(deviceMatrixDesc_t<double> desc, double* A, double* V, hipStream_t stream);
template int check_info(deviceMatrixDesc_t<double> A, const long long);
// float
template void createMatrixDesc(deviceMatrixDesc_t<float>* desc, long long bdim, long long rank, deviceMatrixDesc_t<float> lower, H2Matrix<float>& matrix); 
template void destroyMatrixDesc(deviceMatrixDesc_t<float> desc);
template void copyDataInMatrixDesc(deviceMatrixDesc_t<float> desc, const float* A, const float* U, hipStream_t stream);
template void copyDataOutMatrixDesc(deviceMatrixDesc_t<float> desc, float* A, float* V, hipStream_t stream);
template int check_info(deviceMatrixDesc_t<float> A, const long long);

struct keysD {
  long long D;
  keysD(long long D) : D(D) {}
  __host__ __device__ bool operator()(thrust::tuple<long long, long long, long long, long long> x) const {
    return D + thrust::get<0>(x) == thrust::get<1>(x);
  }
};

template<class T> struct setDevicePtr {
  T* data;
  long long ldx, ldy, ldz;
  setDevicePtr(T* data, long long ldx, long long ldy = 0, long long ldz = 0) : 
    data(data), ldx(ldx), ldy(ldy), ldz(ldz) {}
  __host__ __device__ T* operator()(long long x) const {
    return data + x * ldx;
  }
  __host__ __device__ T* operator()(long long y, long long x) const {
    return data + (x * ldx + y * ldy);
  }
  __host__ __device__ T* operator()(thrust::tuple<long long, long long, long long> x) const {
    return data + (thrust::get<0>(x) * ldx + thrust::get<1>(x) * ldy + thrust::get<2>(x) * ldz);
  }
};

template <typename DT>
void fill_one(deviceMatrixDesc_t<DT>* desc) {
  thrust::fill(thrust::device_ptr<DT>(desc->ONEdata), thrust::device_ptr<DT>(&(desc->ONEdata)[desc->reducLen]), 1.);
}

template <>
void fill_one(deviceMatrixDesc_t<std::complex<double>>* desc) {
  thrust::fill(thrust::device_ptr<hipDoubleComplex>(desc->ONEdata), thrust::device_ptr<hipDoubleComplex>(&(desc->ONEdata)[desc->reducLen]), make_hipDoubleComplex(1., 0.));
}

template <>
void fill_one(deviceMatrixDesc_t<std::complex<float>>* desc) {
  thrust::fill(thrust::device_ptr<hipComplex>(desc->ONEdata), thrust::device_ptr<hipComplex>(&(desc->ONEdata)[desc->reducLen]), make_hipComplex(1., 0.));
}

template <typename DT>
void createMatrixDesc(deviceMatrixDesc_t<DT>* desc, long long bdim, long long rank, deviceMatrixDesc_t<DT> lower, H2Matrix<DT>& matrix) {
  typedef typename deviceMatrixDesc_t<DT>::CT CT;
  desc->bdim = bdim;
  desc->rank = rank;
  //desc->diag_offset = comm.oLocal();
  //desc->lower_offset = (comm.LowerX + lower.diag_offset) * lower.rank;
  //long long lenA = desc->lenA = comm.ARowOffsets.back();
  long long lenA = desc->lenA = matrix.ARows.back();
  //long long M = desc->lenM = comm.lenLocal();
  //long long N = desc->lenN = comm.lenNeighbors();
  long long M = desc->lenM = matrix.nodes;
  long long N = desc->lenN = matrix.nodes;

  //thrust::device_vector<long long> ARowOffset(comm.ARowOffsets.begin(), comm.ARowOffsets.end());
  thrust::device_vector<long long> ARowOffset(matrix.ARows.begin(), matrix.ARows.end());
  thrust::device_vector<long long> ARows(lenA, 0ll);
  //thrust::device_vector<long long> ACols(comm.AColumns.begin(), comm.AColumns.end());
  thrust::device_vector<long long> ACols(matrix.ACols.begin(), matrix.ACols.end());
  thrust::device_vector<long long> ADistCols(lenA);
  thrust::device_vector<long long> AInd(lenA);
  
  auto one_iter = thrust::make_constant_iterator(1ll);
  auto A_iter = thrust::make_zip_iterator(ARows.begin(), ACols.begin(), ADistCols.begin(), AInd.begin());
  thrust::scatter(one_iter, one_iter + (M - 1), ARowOffset.begin() + 1, ARows.begin()); 
  thrust::inclusive_scan(ARows.begin(), ARows.end(), ARows.begin());
  thrust::exclusive_scan_by_key(ARows.begin(), ARows.end(), one_iter, ADistCols.begin(), 0ll);

  thrust::sequence(AInd.begin(), AInd.end(), 0);
  thrust::stable_partition(A_iter, A_iter + lenA, keysD(desc->diag_offset));

  desc->reducLen = 1ll + thrust::reduce(ADistCols.begin(), ADistCols.end(), 0ll, thrust::maximum<long long>());
  //long long lenLA = comm.LowerIndA.size();
  long long lenLA = matrix.LowerIndA.size();
  //const thrust::tuple<long long, long long, long long>* commLA = reinterpret_cast<const thrust::tuple<long long, long long, long long>*>(comm.LowerIndA.data());
  const thrust::tuple<long long, long long, long long>* commLA = reinterpret_cast<const thrust::tuple<long long, long long, long long>*>(matrix.LowerIndA.data());
  thrust::device_vector<thrust::tuple<long long, long long, long long>> LInd(commLA, commLA + lenLA);

  hipMalloc(reinterpret_cast<void**>(&desc->A_ss), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->A_sr), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->A_rs), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->A_rr), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->A_sr_rows), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->A_dst), lenLA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->A_unsort), lenA * sizeof(CT*));

  hipMalloc(reinterpret_cast<void**>(&desc->U_cols), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->U_R), M * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->V_rows), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->V_R), M * sizeof(CT*));

  hipMalloc(reinterpret_cast<void**>(&desc->B_ind), N * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->B_cols), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->B_R), lenA * sizeof(CT*));

  hipMalloc(reinterpret_cast<void**>(&desc->X_cols), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->Y_R_cols), lenA * sizeof(CT*));

  hipMalloc(reinterpret_cast<void**>(&desc->AC_X), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->AC_X_R), lenA * sizeof(CT*));
  hipMalloc(reinterpret_cast<void**>(&desc->AC_ind), lenA * sizeof(CT*));

  long long block = bdim * bdim;
  long long rblock = rank * rank;
  long long acc_len = desc->reducLen * M * std::max(rblock, bdim);

  hipMalloc(reinterpret_cast<void**>(&desc->Adata), lenA * block * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->Udata), N * block * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->Vdata), M * block * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->Bdata), N * block * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->ACdata), acc_len * sizeof(CT));

  hipMalloc(reinterpret_cast<void**>(&desc->Xdata), N * bdim * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->Ydata), N * bdim * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->ONEdata), desc->reducLen * sizeof(CT));
  hipMalloc(reinterpret_cast<void**>(&desc->Ipiv), M * bdim * sizeof(int));
  hipMalloc(reinterpret_cast<void**>(&desc->Info), M * sizeof(int));

  auto inc_iter = thrust::make_counting_iterator(0ll);
  auto rwise_diag_iter = thrust::make_permutation_iterator(AInd.begin(), ARows.begin());
  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);

  thrust::transform(AInd.begin(), AInd.end(), thrust::device_ptr<CT*>(desc->A_ss), setDevicePtr(desc->Adata, block));
  thrust::transform(AInd.begin(), AInd.end(), thrust::device_ptr<CT*>(desc->A_sr), setDevicePtr(&(desc->Adata)[offset_SR], block));
  thrust::transform(AInd.begin(), AInd.end(), thrust::device_ptr<CT*>(desc->A_rs), setDevicePtr(&(desc->Adata)[offset_RS], block));
  thrust::transform(AInd.begin(), AInd.end(), thrust::device_ptr<CT*>(desc->A_rr), setDevicePtr(&(desc->Adata)[offset_RR], block));
  thrust::transform(rwise_diag_iter, rwise_diag_iter + lenA, thrust::device_ptr<CT*>(desc->A_sr_rows), setDevicePtr(&(desc->Adata)[offset_SR], block));
  thrust::transform(LInd.begin(), LInd.end(), thrust::device_ptr<CT*>(desc->A_dst), setDevicePtr(desc->Adata, block, bdim * lower.rank, lower.rank));
  thrust::transform(inc_iter, inc_iter + lenA, thrust::device_ptr<const CT*>(desc->A_unsort), setDevicePtr(desc->Adata, block));

  thrust::transform(ACols.begin(), ACols.end(), thrust::device_ptr<CT*>(desc->U_cols), setDevicePtr(desc->Udata, block));
  thrust::transform(ACols.begin(), ACols.begin() + M, thrust::device_ptr<CT*>(desc->U_R), setDevicePtr(&(desc->Udata)[offset_SR], block));
  thrust::transform(ARows.begin(), ARows.end(), thrust::device_ptr<CT*>(desc->V_rows), setDevicePtr(desc->Vdata, block));
  thrust::transform(inc_iter, inc_iter + M, thrust::device_ptr<CT*>(desc->V_R), setDevicePtr(&(desc->Vdata)[offset_RS], block));

  thrust::transform(inc_iter, inc_iter + N, thrust::device_ptr<CT*>(desc->B_ind), setDevicePtr(desc->Bdata, block));
  thrust::transform(ACols.begin(), ACols.end(), thrust::device_ptr<CT*>(desc->B_cols), setDevicePtr(desc->Bdata, block));
  thrust::transform(ACols.begin(), ACols.end(), thrust::device_ptr<CT*>(desc->B_R), setDevicePtr(&(desc->Bdata)[offset_SR], block));

  thrust::transform(ACols.begin(), ACols.end(), thrust::device_ptr<CT*>(desc->X_cols), setDevicePtr(desc->Xdata, rank));
  thrust::transform(ACols.begin(), ACols.end(), thrust::device_ptr<CT*>(desc->Y_R_cols), setDevicePtr(&(desc->Ydata)[offset_RS], bdim));

  thrust::transform(ARows.begin(), ARows.end(), ADistCols.begin(), thrust::device_ptr<CT*>(desc->AC_X), setDevicePtr(desc->ACdata, M * rank, rank));
  thrust::transform(ARows.begin(), ARows.end(), ADistCols.begin(), thrust::device_ptr<CT*>(desc->AC_X_R), setDevicePtr(&(desc->ACdata)[offset_RS], M * bdim, bdim));
  thrust::transform(ARows.begin(), ARows.end(), ADistCols.begin(), thrust::device_ptr<CT*>(desc->AC_ind), setDevicePtr(desc->ACdata, M * rblock, rblock));
  
  fill_one(desc); 
  //thrust::fill(thrust::device_ptr<CT>(desc->ONEdata), thrust::device_ptr<CT>(&(desc->ONEdata)[desc->reducLen]), make_hipDoubleComplex(1., 0.));

  /*desc->Neighbor = reinterpret_cast<long long*>(std::malloc(comm.BoxOffsets.size() * sizeof(long long)));
  std::copy(comm.BoxOffsets.begin(), comm.BoxOffsets.end(), desc->Neighbor);

  desc->LenComms = comm.NeighborComm.size();
  if (desc->LenComms) {
    desc->NeighborRoots = reinterpret_cast<long long*>(std::malloc(desc->LenComms * sizeof(long long)));
    desc->NeighborComms = reinterpret_cast<ncclComm_t*>(std::malloc(desc->LenComms * sizeof(ncclComm_t)));

    std::transform(comm.NeighborComm.begin(), comm.NeighborComm.end(), desc->NeighborRoots, 
      [](const std::pair<int, MPI_Comm>& comm) { return static_cast<long long>(comm.first); });
    std::transform(comm.NeighborComm.begin(), comm.NeighborComm.end(), desc->NeighborComms, 
      [=](const std::pair<int, MPI_Comm>& comm) { return findNcclComm(comm.second, nccl_comms); });
  }

  desc->DupComm = findNcclComm(comm.DupComm, nccl_comms);
  desc->MergeComm = findNcclComm(comm.MergeComm, nccl_comms);*/
}

template <typename DT>
void destroyMatrixDesc(deviceMatrixDesc_t<DT> desc) {
  hipFree(desc.A_ss);
  hipFree(desc.A_sr);
  hipFree(desc.A_rs);
  hipFree(desc.A_rr);
  hipFree(desc.A_sr_rows);
  hipFree(desc.A_dst);
  hipFree(desc.A_unsort);

  hipFree(desc.U_cols);
  hipFree(desc.U_R);
  hipFree(desc.V_rows);
  hipFree(desc.V_R);

  hipFree(desc.B_ind);
  hipFree(desc.B_cols);
  hipFree(desc.B_R);
  hipFree(desc.AC_ind);

  hipFree(desc.X_cols);
  hipFree(desc.Y_R_cols);
  hipFree(desc.AC_X);
  hipFree(desc.AC_X_R);

  hipFree(desc.Adata);
  hipFree(desc.Udata);
  hipFree(desc.Vdata);
  hipFree(desc.Bdata);
  hipFree(desc.ACdata);

  hipFree(desc.Xdata);
  hipFree(desc.Ydata);
  hipFree(desc.ONEdata);
  hipFree(desc.Ipiv);
  hipFree(desc.Info);

  if (desc.LenComms) {
    std::free(desc.NeighborRoots);
    std::free(desc.NeighborComms);
  }
}

template <typename DT>
void copyDataInMatrixDesc(deviceMatrixDesc_t<DT> desc, const DT* A, const DT* U, hipStream_t stream) {
  long long block = desc.bdim * desc.bdim * sizeof(typename deviceMatrixDesc_t<DT>::CT);
  hipMemcpyAsync(desc.Adata, A, block * desc.lenA, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(desc.Udata, U, block * desc.lenN, hipMemcpyHostToDevice, stream);
}

template <typename DT>
void copyDataOutMatrixDesc(deviceMatrixDesc_t<DT> desc, DT* A, DT* V, hipStream_t stream) {
  long long block = desc.bdim * desc.bdim * sizeof(typename deviceMatrixDesc_t<DT>::CT);
  hipMemcpyAsync(A, desc.Adata, block * desc.lenA, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(V, desc.Vdata, block * desc.lenM, hipMemcpyDeviceToHost, stream);
}

template <typename DT>
int check_info(deviceMatrixDesc_t<DT> A, const long long M) {
  thrust::device_ptr<int> info_ptr(A.Info);
  int sum = thrust::inner_product(info_ptr, info_ptr + M, info_ptr, 0);
  return 0 < sum;
}
