#include "hip/hip_runtime.h"

#include <device_csr_matrix.cuh>
#include <device_factorize.cuh>
#include <comm-mpi.hpp>

#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/inner_product.h>

#include <iostream>

struct conjugateFunc {
  __host__ __device__ thrust::complex<float> operator()(const thrust::complex<float>& z) const {
    return thrust::conj(z);
  }
  __host__ __device__ thrust::complex<double> operator()(const thrust::complex<double>& z) const {
    return thrust::conj(z);
  }
};

void levelCommSum(long long N, thrust::complex<double> X[], ncclComm_t AllComm, ncclComm_t DupComm, hipStream_t stream) {
  if (AllComm)
    ncclAllReduce(const_cast<const thrust::complex<double>*>(X), X, N * 2, ncclDouble, ncclSum, AllComm, stream);
  if (DupComm)
    ncclBroadcast(const_cast<const thrust::complex<double>*>(X), X, N * 2, ncclDouble, 0, DupComm, stream);
}

void levelCommSum(long long N, thrust::complex<float> X[], ncclComm_t AllComm, ncclComm_t DupComm, hipStream_t stream) {
  if (AllComm)
    ncclAllReduce(const_cast<const thrust::complex<float>*>(X), X, N * 2, ncclFloat, ncclSum, AllComm, stream);
  if (DupComm)
    ncclBroadcast(const_cast<const thrust::complex<float>*>(X), X, N * 2, ncclFloat, 0, DupComm, stream);
}

template <>
long long solveDeviceGMRES(deviceHandle_t handle, long long levels, CsrMatVecDesc_t<std::complex<double>> desc[], long long mlevels, deviceMatrixDesc_t<std::complex<double>> desc_m[], double tol, std::complex<double>* X, const std::complex<double>* B, long long inner_iters, long long outer_iters, double resid[], const ColCommMPI& comm, const ncclComms nccl_comms) {
  long long N = desc[levels]->X->lenX;
  long long ld = inner_iters + 1;

  thrust::device_vector<thrust::complex<double>> devB(B, &B[N]);
  thrust::device_vector<thrust::complex<double>> devR(devB.begin(), devB.end());
  thrust::device_vector<thrust::complex<double>> devX(N, thrust::complex<double>(0., 0.));
  thrust::device_vector<thrust::complex<double>> H(ld * inner_iters);
  thrust::device_vector<thrust::complex<double>> v(N * ld);
  thrust::device_vector<thrust::complex<double>> s(ld);
  thrust::device_vector<thrust::complex<double>*> ptr({thrust::raw_pointer_cast(H.data()), thrust::raw_pointer_cast(s.data())});

  hipDoubleComplex* Bdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(devB.data()));
  hipDoubleComplex* Rdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(devR.data()));
  hipDoubleComplex* Xdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(devX.data()));
  hipDoubleComplex* Hdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(H.data()));
  hipDoubleComplex* Vdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(v.data()));
  hipDoubleComplex* Sdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(s.data()));
  hipDoubleComplex** Pdata = reinterpret_cast<hipDoubleComplex**>(thrust::raw_pointer_cast(ptr.data()));

  ncclComm_t AllComm = findNcclComm(comm.AllReduceComm, nccl_comms);
  ncclComm_t DupComm = findNcclComm(comm.DupComm, nccl_comms);
  hipStream_t stream = handle->compute_stream;
  hipblasHandle_t cublasH = handle->cublasH;
  int* dev_info;
  hipMalloc(reinterpret_cast<void**>(&dev_info), sizeof(int));

  auto conjR = thrust::make_transform_iterator(devR.begin(), conjugateFunc());
  thrust::complex<double> nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
    thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
  comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);

  double normb = std::sqrt(nsum.real());
  if (normb == 0.)
    normb = 1.;
  resid[0] = 1.;
  long long iters = 0;

  thrust::complex<double> one(1., 0.), zero(0., 0.), minus_one(-1., 0.);

  while (iters < outer_iters && tol <= resid[iters]) {
    matSolvePreconditionDeviceH2(handle, mlevels, desc_m, reinterpret_cast<std::complex<double>*>(Rdata));
    nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
      thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
    comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);

    double beta = std::sqrt(nsum.real());
    thrust::complex<double> inv_beta(1. / beta, 0.);
    thrust::fill(H.begin(), H.end(), thrust::complex<double>(0., 0.));
    thrust::fill(v.begin(), v.end(), thrust::complex<double>(0., 0.));
    thrust::fill(s.begin() + 1, s.end(), zero);
    s[0] = beta;
    hipblasZaxpy(cublasH, N, reinterpret_cast<hipDoubleComplex*>(&inv_beta), Rdata, 1, Vdata, 1);
    
    for (long long i = 0; i < inner_iters; i++) {
      hipblasZcopy(cublasH, N, &Vdata[i * N], 1, Rdata, 1);
      matVecDeviceH2(handle, levels, desc, reinterpret_cast<std::complex<double>*>(Rdata));
      matSolvePreconditionDeviceH2(handle, mlevels, desc_m, reinterpret_cast<std::complex<double>*>(Rdata));

      hipblasZgemv(cublasH, HIPBLAS_OP_C, N, i + 1, reinterpret_cast<hipDoubleComplex*>(&one), Vdata, N, Rdata, 1, reinterpret_cast<hipDoubleComplex*>(&zero), &Hdata[i * ld], 1);
      levelCommSum(i + 1, thrust::raw_pointer_cast(&H[i * ld]), AllComm, DupComm, stream);
      hipblasZgemv(cublasH, HIPBLAS_OP_N, N, i + 1, reinterpret_cast<hipDoubleComplex*>(&minus_one), Vdata, N, &Hdata[i * ld], 1, reinterpret_cast<hipDoubleComplex*>(&one), Rdata, 1);

      nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
        thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
      comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);

      H[i * (ld + 1) + 1] = std::sqrt(nsum.real());
      thrust::complex<double> inv_beta(1. / std::sqrt(nsum.real()), 0.);
      hipblasZaxpy(cublasH, N, reinterpret_cast<hipDoubleComplex*>(&inv_beta), Rdata, 1, &Vdata[N * (i + 1)], 1);
    }
    
    int info;
    hipblasZgelsBatched(cublasH, HIPBLAS_OP_N, ld, inner_iters, 1, Pdata, ld, &Pdata[1], ld, &info, dev_info, 1);
    hipblasZgemv(cublasH, HIPBLAS_OP_N, N, inner_iters, reinterpret_cast<hipDoubleComplex*>(&one), Vdata, N, Sdata, 1, reinterpret_cast<hipDoubleComplex*>(&one), Xdata, 1);

    hipblasZgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1, reinterpret_cast<hipDoubleComplex*>(&minus_one), Xdata, N, reinterpret_cast<hipDoubleComplex*>(&zero), Rdata, N, Rdata, N);
    matVecDeviceH2(handle, levels, desc, reinterpret_cast<std::complex<double>*>(Rdata));
    hipblasZaxpy(cublasH, N, reinterpret_cast<hipDoubleComplex*>(&one), Bdata, 1, Rdata, 1);

    nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
      thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
    comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);
    resid[++iters] = std::sqrt(nsum.real()) / normb;
  }

  thrust::copy(devX.begin(), devX.end(), X);
  hipFree(dev_info);
  return iters;
}

template <>
long long solveDeviceGMRES(deviceHandle_t handle, long long levels, CsrMatVecDesc_t<std::complex<double>> desc[], long long mlevels, deviceMatrixDesc_t<std::complex<float>> desc_m[], double tol, std::complex<double>* X, const std::complex<double>* B, long long inner_iters, long long outer_iters, double resid[], const ColCommMPI& comm, const ncclComms nccl_comms) {
  long long N = desc[levels]->X->lenX;
  long long ld = inner_iters + 1;

  thrust::device_vector<thrust::complex<double>> devB(B, &B[N]);
  thrust::device_vector<thrust::complex<double>> devR(devB.begin(), devB.end());
  thrust::device_vector<thrust::complex<float>> devR_low(N);
  thrust::device_vector<thrust::complex<double>> devX(N, thrust::complex<double>(0., 0.));
  thrust::device_vector<thrust::complex<double>> H(ld * inner_iters);
  thrust::device_vector<thrust::complex<double>> v(N * ld);
  thrust::device_vector<thrust::complex<double>> s(ld);
  thrust::device_vector<thrust::complex<double>*> ptr({thrust::raw_pointer_cast(H.data()), thrust::raw_pointer_cast(s.data())});

  hipDoubleComplex* Bdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(devB.data()));
  hipDoubleComplex* Rdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(devR.data()));
  hipComplex* Rdata_low = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(devR_low.data()));
  hipDoubleComplex* Xdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(devX.data()));
  hipDoubleComplex* Hdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(H.data()));
  hipDoubleComplex* Vdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(v.data()));
  hipDoubleComplex* Sdata = reinterpret_cast<hipDoubleComplex*>(thrust::raw_pointer_cast(s.data()));
  hipDoubleComplex** Pdata = reinterpret_cast<hipDoubleComplex**>(thrust::raw_pointer_cast(ptr.data()));

  ncclComm_t AllComm = findNcclComm(comm.AllReduceComm, nccl_comms);
  ncclComm_t DupComm = findNcclComm(comm.DupComm, nccl_comms);
  hipStream_t stream = handle->compute_stream;
  hipblasHandle_t cublasH = handle->cublasH;
  int* dev_info;
  hipMalloc(reinterpret_cast<void**>(&dev_info), sizeof(int));

  auto conjR = thrust::make_transform_iterator(devR.begin(), conjugateFunc());
  thrust::complex<double> nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
    thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
  comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);

  double normb = std::sqrt(nsum.real());
  if (normb == 0.)
    normb = 1.;
  resid[0] = 1.;
  long long iters = 0;

  thrust::complex<double> one(1., 0.), zero(0., 0.), minus_one(-1., 0.);

  while (iters < outer_iters && tol <= resid[iters]) {
    devR_low = devR;
    matSolvePreconditionDeviceH2(handle, mlevels, desc_m, reinterpret_cast<std::complex<float>*>(Rdata_low));
    devR = devR_low;
    nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
      thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
    comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);

    double beta = std::sqrt(nsum.real());
    thrust::complex<double> inv_beta(1. / beta, 0.);
    thrust::fill(H.begin(), H.end(), thrust::complex<double>(0., 0.));
    thrust::fill(v.begin(), v.end(), thrust::complex<double>(0., 0.));
    thrust::fill(s.begin() + 1, s.end(), zero);
    s[0] = beta;
    hipblasZaxpy(cublasH, N, reinterpret_cast<hipDoubleComplex*>(&inv_beta), Rdata, 1, Vdata, 1);
    
    for (long long i = 0; i < inner_iters; i++) {
      hipblasZcopy(cublasH, N, &Vdata[i * N], 1, Rdata, 1);
      matVecDeviceH2(handle, levels, desc, reinterpret_cast<std::complex<double>*>(Rdata));
      devR_low = devR;
      matSolvePreconditionDeviceH2(handle, mlevels, desc_m, reinterpret_cast<std::complex<float>*>(Rdata_low));
      devR = devR_low;

      hipblasZgemv(cublasH, HIPBLAS_OP_C, N, i + 1, reinterpret_cast<hipDoubleComplex*>(&one), Vdata, N, Rdata, 1, reinterpret_cast<hipDoubleComplex*>(&zero), &Hdata[i * ld], 1);
      levelCommSum(i + 1, thrust::raw_pointer_cast(&H[i * ld]), AllComm, DupComm, stream);
      hipblasZgemv(cublasH, HIPBLAS_OP_N, N, i + 1, reinterpret_cast<hipDoubleComplex*>(&minus_one), Vdata, N, &Hdata[i * ld], 1, reinterpret_cast<hipDoubleComplex*>(&one), Rdata, 1);

      nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
        thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
      comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);

      H[i * (ld + 1) + 1] = std::sqrt(nsum.real());
      thrust::complex<double> inv_beta(1. / std::sqrt(nsum.real()), 0.);
      hipblasZaxpy(cublasH, N, reinterpret_cast<hipDoubleComplex*>(&inv_beta), Rdata, 1, &Vdata[N * (i + 1)], 1);
    }
    
    int info;
    hipblasZgelsBatched(cublasH, HIPBLAS_OP_N, ld, inner_iters, 1, Pdata, ld, &Pdata[1], ld, &info, dev_info, 1);
    hipblasZgemv(cublasH, HIPBLAS_OP_N, N, inner_iters, reinterpret_cast<hipDoubleComplex*>(&one), Vdata, N, Sdata, 1, reinterpret_cast<hipDoubleComplex*>(&one), Xdata, 1);

    hipblasZgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1, reinterpret_cast<hipDoubleComplex*>(&minus_one), Xdata, N, reinterpret_cast<hipDoubleComplex*>(&zero), Rdata, N, Rdata, N);
    matVecDeviceH2(handle, levels, desc, reinterpret_cast<std::complex<double>*>(Rdata));
    hipblasZaxpy(cublasH, N, reinterpret_cast<hipDoubleComplex*>(&one), Bdata, 1, Rdata, 1);

    nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
      thrust::complex<double>(0., 0.), thrust::plus<thrust::complex<double>>(), thrust::multiplies<thrust::complex<double>>());
    comm.level_sum(reinterpret_cast<std::complex<double>*>(&nsum), 1);
    resid[++iters] = std::sqrt(nsum.real()) / normb;
  }

  thrust::copy(devX.begin(), devX.end(), X);
  hipFree(dev_info);
  return iters;
}

template <>
long long solveDeviceGMRES(deviceHandle_t handle, long long levels, CsrMatVecDesc_t<std::complex<float>> desc[], long long mlevels, deviceMatrixDesc_t<std::complex<float>> desc_m[], double tol, std::complex<float>* X, const std::complex<float>* B, long long inner_iters, long long outer_iters, double resid[], const ColCommMPI& comm, const ncclComms nccl_comms) {
    long long N = desc[levels]->X->lenX;
  long long ld = inner_iters + 1;

  thrust::device_vector<thrust::complex<float>> devB(B, &B[N]);
  thrust::device_vector<thrust::complex<float>> devR(devB.begin(), devB.end());
  thrust::device_vector<thrust::complex<float>> devX(N, thrust::complex<float>(0., 0.));
  thrust::device_vector<thrust::complex<float>> H(ld * inner_iters);
  thrust::device_vector<thrust::complex<float>> v(N * ld);
  thrust::device_vector<thrust::complex<float>> s(ld);
  thrust::device_vector<thrust::complex<float>*> ptr({thrust::raw_pointer_cast(H.data()), thrust::raw_pointer_cast(s.data())});

  hipComplex* Bdata = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(devB.data()));
  hipComplex* Rdata = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(devR.data()));
  hipComplex* Xdata = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(devX.data()));
  hipComplex* Hdata = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(H.data()));
  hipComplex* Vdata = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(v.data()));
  hipComplex* Sdata = reinterpret_cast<hipComplex*>(thrust::raw_pointer_cast(s.data()));
  hipComplex** Pdata = reinterpret_cast<hipComplex**>(thrust::raw_pointer_cast(ptr.data()));

  ncclComm_t AllComm = findNcclComm(comm.AllReduceComm, nccl_comms);
  ncclComm_t DupComm = findNcclComm(comm.DupComm, nccl_comms);
  hipStream_t stream = handle->compute_stream;
  hipblasHandle_t cublasH = handle->cublasH;
  int* dev_info;
  hipMalloc(reinterpret_cast<void**>(&dev_info), sizeof(int));

  auto conjR = thrust::make_transform_iterator(devR.begin(), conjugateFunc());
  thrust::complex<float> nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
    thrust::complex<float>(0., 0.), thrust::plus<thrust::complex<float>>(), thrust::multiplies<thrust::complex<float>>());
  comm.level_sum(reinterpret_cast<std::complex<float>*>(&nsum), 1);

  float normb = std::sqrt(nsum.real());
  if (normb == 0.)
    normb = 1.;
  resid[0] = 1.;
  long long iters = 0;

  thrust::complex<float> one(1., 0.), zero(0., 0.), minus_one(-1., 0.);

  while (iters < outer_iters && tol <= resid[iters]) {
    matSolvePreconditionDeviceH2(handle, mlevels, desc_m, reinterpret_cast<std::complex<float>*>(Rdata));
    nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
      thrust::complex<float>(0., 0.), thrust::plus<thrust::complex<float>>(), thrust::multiplies<thrust::complex<float>>());
    comm.level_sum(reinterpret_cast<std::complex<float>*>(&nsum), 1);

    float beta = std::sqrt(nsum.real());
    thrust::complex<float> inv_beta(1. / beta, 0.);
    thrust::fill(H.begin(), H.end(), thrust::complex<float>(0., 0.));
    thrust::fill(v.begin(), v.end(), thrust::complex<float>(0., 0.));
    thrust::fill(s.begin() + 1, s.end(), zero);
    s[0] = beta;
    hipblasCaxpy(cublasH, N, reinterpret_cast<hipComplex*>(&inv_beta), Rdata, 1, Vdata, 1);
    
    for (long long i = 0; i < inner_iters; i++) {
      hipblasCcopy(cublasH, N, &Vdata[i * N], 1, Rdata, 1);
      matVecDeviceH2(handle, levels, desc, reinterpret_cast<std::complex<float>*>(Rdata));
      matSolvePreconditionDeviceH2(handle, mlevels, desc_m, reinterpret_cast<std::complex<float>*>(Rdata));

      hipblasCgemv(cublasH, HIPBLAS_OP_C, N, i + 1, reinterpret_cast<hipComplex*>(&one), Vdata, N, Rdata, 1, reinterpret_cast<hipComplex*>(&zero), &Hdata[i * ld], 1);
      levelCommSum(i + 1, thrust::raw_pointer_cast(&H[i * ld]), AllComm, DupComm, stream);
      hipblasCgemv(cublasH, HIPBLAS_OP_N, N, i + 1, reinterpret_cast<hipComplex*>(&minus_one), Vdata, N, &Hdata[i * ld], 1, reinterpret_cast<hipComplex*>(&one), Rdata, 1);

      nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
        thrust::complex<float>(0., 0.), thrust::plus<thrust::complex<float>>(), thrust::multiplies<thrust::complex<float>>());
      comm.level_sum(reinterpret_cast<std::complex<float>*>(&nsum), 1);

      H[i * (ld + 1) + 1] = std::sqrt(nsum.real());
      thrust::complex<float> inv_beta(1. / std::sqrt(nsum.real()), 0.);
      hipblasCaxpy(cublasH, N, reinterpret_cast<hipComplex*>(&inv_beta), Rdata, 1, &Vdata[N * (i + 1)], 1);
    }
    
    int info;
    hipblasCgelsBatched(cublasH, HIPBLAS_OP_N, ld, inner_iters, 1, Pdata, ld, &Pdata[1], ld, &info, dev_info, 1);
    hipblasCgemv(cublasH, HIPBLAS_OP_N, N, inner_iters, reinterpret_cast<hipComplex*>(&one), Vdata, N, Sdata, 1, reinterpret_cast<hipComplex*>(&one), Xdata, 1);

    hipblasCgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1, reinterpret_cast<hipComplex*>(&minus_one), Xdata, N, reinterpret_cast<hipComplex*>(&zero), Rdata, N, Rdata, N);
    matVecDeviceH2(handle, levels, desc, reinterpret_cast<std::complex<float>*>(Rdata));
    hipblasCaxpy(cublasH, N, reinterpret_cast<hipComplex*>(&one), Bdata, 1, Rdata, 1);

    nsum = thrust::inner_product(thrust::cuda::par.on(stream), conjR, conjR + N, devR.begin(), 
      thrust::complex<float>(0., 0.), thrust::plus<thrust::complex<float>>(), thrust::multiplies<thrust::complex<float>>());
    comm.level_sum(reinterpret_cast<std::complex<float>*>(&nsum), 1);
    resid[++iters] = std::sqrt(nsum.real()) / normb;
  }

  thrust::copy(devX.begin(), devX.end(), X);
  hipFree(dev_info);
  return iters;
}

template <>
long long solveDeviceGMRES(deviceHandle_t handle, long long levels, CsrMatVecDesc_t<double> desc[], long long mlevels, deviceMatrixDesc_t<double> desc_m[], double tol, double* X, const double* B, long long inner_iters, long long outer_iters, double resid[], const ColCommMPI& comm, const ncclComms nccl_comms) {
  std::cout<<"Not implemented"<<std::endl;
}

template <>
long long solveDeviceGMRES(deviceHandle_t handle, long long levels, CsrMatVecDesc_t<float> desc[], long long mlevels, deviceMatrixDesc_t<float> desc_m[], double tol, float* X, const float* B, long long inner_iters, long long outer_iters, double resid[], const ColCommMPI& comm, const ncclComms nccl_comms) {
  std::cout<<"Not implemented"<<std::endl;
}
